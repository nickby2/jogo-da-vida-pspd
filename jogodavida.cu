#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ind2d(i, j, tam) ((i) * (tam + 2) + (j))

__global__ void UmaVidaKernel(int* tabulIn, int* tabulOut, int tam) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (i <= tam && j <= tam) {
        int vizviv = tabulIn[ind2d(i-1, j-1, tam)] + tabulIn[ind2d(i-1, j  , tam)] +
                     tabulIn[ind2d(i-1, j+1, tam)] + tabulIn[ind2d(i  , j-1, tam)] +
                     tabulIn[ind2d(i  , j+1, tam)] + tabulIn[ind2d(i+1, j-1, tam)] +
                     tabulIn[ind2d(i+1, j  , tam)] + tabulIn[ind2d(i+1, j+1, tam)];

        if (tabulIn[ind2d(i, j, tam)] && vizviv < 2)
            tabulOut[ind2d(i, j, tam)] = 0;
        else if (tabulIn[ind2d(i, j, tam)] && vizviv > 3)
            tabulOut[ind2d(i, j, tam)] = 0;
        else if (!tabulIn[ind2d(i, j, tam)] && vizviv == 3)
            tabulOut[ind2d(i, j, tam)] = 1;
        else
            tabulOut[ind2d(i, j, tam)] = tabulIn[ind2d(i, j, tam)];
    }
}

void UmaVida(int* d_in, int* d_out, int tam) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((tam + 15) / 16, (tam + 15) / 16);
    UmaVidaKernel<<<numBlocks, threadsPerBlock>>>(d_in, d_out, tam);
    hipDeviceSynchronize();
}

void InicializaTabuleiro(int* tabul, int tam) {
    for (int i = 0; i < (tam + 2) * (tam + 2); i++) {
        tabul[i] = rand() % 2;
    }
}

int main() {
    int tam = 32;
    int maxGen = 100;

    size_t size = (tam + 2) * (tam + 2) * sizeof(int);

    int* h_tabul1 = (int*)malloc(size);
    int* h_tabul2 = (int*)malloc(size);

    InicializaTabuleiro(h_tabul1, tam);

    int* d_tabul1;
    int* d_tabul2;

    hipMalloc((void**)&d_tabul1, size);
    hipMalloc((void**)&d_tabul2, size);

    hipMemcpy(d_tabul1, h_tabul1, size, hipMemcpyHostToDevice);

    FILE* arquivo = fopen("geracoes.txt", "w");

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    for (int gen = 0; gen < maxGen; gen++) {
        UmaVida(d_tabul1, d_tabul2, tam);

        hipMemcpy(h_tabul1, d_tabul1, size, hipMemcpyDeviceToHost);

        fprintf(arquivo, "Geração %d:\n", gen + 1);
        for (int i = 1; i <= tam; i++) {
            for (int j = 1; j <= tam; j++) {
                fprintf(arquivo, "%d ", h_tabul1[ind2d(i, j, tam)]);
            }
            fprintf(arquivo, "\n");
        }
        fprintf(arquivo, "\n");

        int* temp = d_tabul1;
        d_tabul1 = d_tabul2;
        d_tabul2 = temp;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_tabul1, d_tabul1, size, hipMemcpyDeviceToHost);

    printf("Simulação completa!\n");
    printf("Tempo de execução na GPU: %.4f ms\n", milliseconds);

    fclose(arquivo);
    hipFree(d_tabul1);
    hipFree(d_tabul2);
    free(h_tabul1);
    free(h_tabul2);

    return 0;
}
